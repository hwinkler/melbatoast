
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int *a, int *b, int *c) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  c[idx] = a[idx] + b[idx];
}

void check (const char* msg, int e){
  printf ("%s: %d\n", msg, e);
}

int main (void){

  const int N = 512; // number of vector elements  
  const int M = 8; // threads per block

  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);

  a = (int*) calloc(N, sizeof(int));
  b = (int*) calloc(N, sizeof(int));
  c = (int*) calloc(N, sizeof(int));
  
  check("hipMalloc d_a", hipMalloc((void**)&d_a, size));
  check("hipMalloc d_b",hipMalloc((void**)&d_b, size));
  check("hipMalloc d_c",hipMalloc((void**)&d_c, size));
  
  for (int i =0; i<N; i++){
    a[i] = i;
    b[i] = N/2 - i;
  }
  
  check("hipMemcpy d_a<-a", hipMemcpy (d_a, a, size, hipMemcpyHostToDevice));
  check("hipMemcpy d_b<-b", hipMemcpy (d_b, b, size, hipMemcpyHostToDevice));

  add<<<N,M>>>(d_a, d_b, d_c);

  check("hipMemcpy c<-d_c", hipMemcpy (c, d_c, size, hipMemcpyDeviceToHost));
  check("hipFree a", hipFree(d_a));
  check("hipFree b", hipFree(d_b));
  check("hipFree c", hipFree(d_c));
  for (int i =0; i<N; i++){
    if (c[i] != a[i] + b[i] ){
      printf ("c[%d]= %d, should be %d\n", i, c[i], a[i] + b[i]);
      return 1;
    }
  }
  printf("ok\n");
  return 0;
}

