
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void check (const char* msg, int e){
  printf ("%s: %d\n", msg, e);
}

int main (void){
  const int N = 512;   
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);

  a = (int*) calloc(N, sizeof(int));
  b = (int*) calloc(N, sizeof(int));
  c = (int*) calloc(N, sizeof(int));
  
  check("hipMalloc d_a", hipMalloc((void**)&d_a, size));
  check("hipMalloc d_b",hipMalloc((void**)&d_b, size));
  check("hipMalloc d_c",hipMalloc((void**)&d_c, size));
  
  for (int i =0; i<N; i++){
    a[i] = i;
    b[i] = N/2 - i;
  }
  
  check("hipMemcpy d_a<-a", hipMemcpy (d_a, a, size, hipMemcpyHostToDevice));
  check("hipMemcpy d_b<-b", hipMemcpy (d_b, b, size, hipMemcpyHostToDevice));

  add<<<N,1>>>(d_a, d_b, d_c);

  check("hipMemcpy c<-d_c", hipMemcpy (c, d_c, size, hipMemcpyDeviceToHost));
  check("hipFree a", hipFree(d_a));
  check("hipFree b", hipFree(d_b));
  check("hipFree c", hipFree(d_c));
  for (int i =0; i<N; i++){
    if (c[i] != a[i] + b[i] ){
      printf ("c[%d]= %d, should be %d\n", i, c[i], a[i] + b[i]);
      return 1;
    }
  }
  printf("ok\n");
  return 0;
}

