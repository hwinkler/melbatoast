
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int *a, int *b, int *c) {
  *c = *a + *b;
}

void check (const char* msg, int e){
  printf ("%s: %d\n", msg, e);
}

int main (void){
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);

  check("hipMalloc a", hipMalloc((void**)&d_a, size));
  check("hipMalloc b",hipMalloc((void**)&d_b, size));
  check("hipMalloc c",hipMalloc((void**)&d_c, size));
  
  a = 2;
  b = 7;
  
  check("hipMemcpy d_a<-a", hipMemcpy (d_a, &a, size, hipMemcpyHostToDevice));
  check("hipMemcpy d_b<-b", hipMemcpy (d_b, &b, size, hipMemcpyHostToDevice));

  add<<<1,1>>>(d_a, d_b, d_c);

  check("hipMemcpy c<-d_c", hipMemcpy (&c, d_c, size, hipMemcpyDeviceToHost));
  check("hipFree a", hipFree(d_a));
  check("hipFree b", hipFree(d_b));
  check("hipFree c", hipFree(d_c));
  printf ("c= %d\n", c);
  return 0;
}

