#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include "potential.h"
#include "gibbs.h"
#include "cudacall.h"

int printDevicePotential (Potential*pd) {
  Potential p;  
  CUDA_CALL(hipMemcpy ( &p,  pd, sizeof(p), hipMemcpyDeviceToHost));

  printf("%17s %6d\n",  "numStates", p.numStates);

  if (p.numConditionals >=0 && p.numConditionals < 1000){
    float conditionals[p.numConditionals];
    CUDA_CALL(hipMemcpy ( conditionals,  p.conditionals, p.numConditionals, hipMemcpyDeviceToHost));
    for (int i=0; i< p.numConditionals; i++){
      printf("%11s[%3d] %6.3f\n",  "conditionals", i,  p.conditionals[i]);
    }
  }
  printf("%17s %6d\n",  "numConditionals", p.numConditionals);
  printf("%17s %6d\n",  "numParents", p.numParents);
  printf("%17s %6d\n",  "numChildren", p.numChildren);
  

  if (p.numParents >=0 && p.numParents <= MAX_PARENTS){
    for (int i=0; i<p.numParents; i++){
      int offset = p.parents[i] - pd;
      printf("%11s[%3d] %6d\n",  "parent", i, offset);
    }
  }
  if (p.numChildren >=0 && p.numChildren <= MAX_CHILDREN){
    for (int i=0; i<p.numChildren; i++){
      int offset = p.children[i] - pd;
      printf("%11s[%3d] %6d\n",  "child", i, offset);
    }
  }
  if (p.numChildren >=0 && p.numChildren <= MAX_CHILDREN){
    for (int i=0; i<p.numChildren; i++){
      printf("%11s[%3d] %6d\n",  "indexInChild", i, p.indexInChild[i]);
    }
  }
  if (p.numParents >=0 && p.numParents <= MAX_PARENTS){
    for (int i=0; i<=p.numParents; i++){
      printf("%11s[%3d] %6d\n",  "dimensions", i, p.dimensions[i]);
    }
  }
  printf("%17s %6d\n",  "isFrozen", p.isFrozen);
  return 0;
}


  __global__ void add(int *a, int *b, int *c);
int main(void) {

  int a, b, c; // host copies of a, b, c 
  int *d_a, *d_b, *d_c; // device copies of a, b, c 
  int size = sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size); 
  hipMalloc((void **)&d_b, size); 
  hipMalloc((void **)&d_c, size);
  // Setup input values
  a = 2; b = 7;
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c); 


  printf("%d + %d = %d\n", a,b,c);
  return 0;
}

int mainXXX (int argc, char ** argv){

  int N = 1;

  const int numPotentials = 5;
  Potential* devPotentials;
  CUDA_CALL(hipMalloc ( (void**) &devPotentials, numPotentials *sizeof( Potential ) ));
  Potential * da = devPotentials+0, *db = devPotentials+1, *dc=devPotentials+2, *dd=devPotentials+3, *de = devPotentials+4;


  const int numConditionals = 2+4+4+4+8 ;
  float conditionals[numConditionals] = {
    // a
    0.4f, 0.6f,
    //b
    0.3f, 0.7f, 0.8f, 0.2f,
    //c
    0.7f, 0.3f, 0.4f, 0.6f,
    //d
    0.5f, 0.5f, 0.1f, 0.9,
    // e
    0.9f, 0.1f, 0.999f,0.001f,
    0.999f, 0.001f,  0.999f,0.001f};

  float *devConditionals;
  CUDA_CALL(hipMalloc ( (void**) &devConditionals, numConditionals * sizeof( float ) ));
  CUDA_CALL(hipMemcpy (devConditionals, conditionals, numConditionals* sizeof(float), hipMemcpyHostToDevice));
  float *dca = devConditionals+0, *dcb = dca + 2, *dcc = dcb +4, *dcd = dcc + 4, *dce = dcd + 4;

  const int numStates[numPotentials] = {2,2,2,2,2};
  // P(A)
  initPotential<<<1, 1>>> (da, numStates[0], dca, 
                           (Potential *[]) {NULL}, 0 );


  // P(B|A)
  initPotential<<<1, 1>>> (db, numStates[1], dcb, 
                           (Potential *[]) {da}, 1 );
  // P(C|A)
  initPotential<<<1, 1>>> (dc, numStates[2], dcc, 
                           (Potential *[]) {da}, 1 );
  // P(D|B)
  initPotential<<<1, 1>>> (dd, numStates[3], dcd, 
                           (Potential *[]) {db}, 1 );
  // P(E|D,C)
  initPotential<<<1, 1>>> (de, numStates[4], dce,
                           (Potential *[]) {dd, dc}, 2 );
  
  
  for (int i=0; i< numPotentials; i++){
    Potential* p = da + i;
    printf ("Potential %c %p:\n", 'A' + i, p);
    printDevicePotential(p);
  }

  //data: B=n, E=n
 
 
  // b->isFrozen = e->isFrozen = true;
  
  int numConfigurations = 1;
  for (int i=0; i< numPotentials; i++){
    numConfigurations *= numStates[i];
  }
  int counts[numConfigurations];
  memset (counts, 0, numConfigurations * sizeof(int));

  // initial config: ynyyn  (we use y=0, n=1)
  int  states [numPotentials] = {0,1,0,0,1};
  int * devStates;
  CUDA_CALL(hipMalloc((void**)&devStates,  numPotentials* sizeof(int)));
  CUDA_CALL(hipMemcpy (devStates, states, numPotentials* sizeof(int), hipMemcpyHostToDevice));

  int * devCounts ;
  CUDA_CALL(hipMalloc( (void**) &devCounts, numConfigurations* sizeof(int)));
  CUDA_CALL(hipMemcpy (devCounts, counts, numConfigurations* sizeof(int), hipMemcpyHostToDevice));

  gibbs<<<1,N>>>(devPotentials, numPotentials, devStates, devCounts, numConfigurations, 100);

  CUDA_CALL(hipMemcpy ( counts,  devCounts, numConfigurations* sizeof(int), hipMemcpyDeviceToHost));

  for (int j=0; j < numConfigurations; j++){
    printf("%4d: %4d\n", j, counts[j]);
  }


  return 0;
}
