#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ double rnd(hiprandState* state) {
  return hiprand_uniform( state );
}

__device__ void rndSeed ( hiprandState * state){
  unsigned int seed = (unsigned int) clock64();
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init ( seed, id, 0, &state[id] );
} 
